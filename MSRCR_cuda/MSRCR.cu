#include "hip/hip_runtime.h"
#include "MSRCR.h"

#define BLK_SZ 16

MSRCR::~MSRCR()
{

}

__global__  void HF_Enhancer_kernel(float *d_out, float *d_in, int wid, int hei)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    //if(idx >= (wid >> 2) || idy >= hei)
    if(idx >= wid || idy >= hei)
        return ;

    // do four calculations per thread
#pragma unloop
    for(int i = 0; i < 4; ++i)   // column direction
    {
        if(idx + i >= wid)
            return;
#pragma unloop
        for(int j = 0; j < 4; ++j)   // row direction
        {
            if(idy + j >= hei)
                return;
            int offset = (j + idy) * wid + (idx + i);

            d_out[offset] = expf(logf(d_out[offset]) - logf(d_in[offset]));
        }
    }

    // do four calculations per thread, but reduce the memory coalescing
    /*
    for(int i = blockDim.x * gridDim.x; i > 0; i = i >> 2)   // divide 4 every time
    {
    }
    */
}

void MSRCR::High_Frequency_Enhancer(float *d_out, float *d_in, int wid, int hei)
{
    //dim3 threadPerBlock(((wid >> 2) + BLK_SZ - 1)/ BLK_SZ, ((hei >> 2) + BLK_SZ - 1) / BLK_SZ);
    dim3 threadPerBlock(BLK_SZ, BLK_SZ);
    dim3 blockPerGrid;
    blockPerGrid.x = ((wid >> 2) + BLK_SZ - 1) / BLK_SZ;
    blockPerGrid.y = ((hei >> 2) + BLK_SZ - 1) / BLK_SZ;

    HF_Enhancer_kernel<<<blockPerGrid, threadPerBlock>>>(d_out, d_in, wid, hei);
}

void MSRCR::MSR(float *d_out, float *d_in, int wid, int hei, double sigma)
{

}

void MSRCR::histEqu(float *d_out, float *d_in, int wid, int hei)
{

}




