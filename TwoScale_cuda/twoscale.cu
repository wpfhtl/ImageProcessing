#include "hip/hip_runtime.h"
#include "twoscale.h"

#define BLK_SZ 16
#define cudaCheckError(err) __cudaCheckError(err, __FILE__, __LINE__)

inline void __cudaCheckError(hipError_t err, const char *file, int line)
{
	if(err != hipSuccess)
	{
		cout << err << " in " << file << " at " << line << " line" << endl;
	}
}

BFilter::BFilter(int wid, int hei)
{
	cudaCheckError(hipMalloc((void**)&d_imgIn_, sizeof(float) * wid * hei));
	cudaCheckError(hipMalloc((void **)&d_imgOut_, sizeof(float) * wid * hei));
}

BFilter::~BFilter()
{
	if(d_imgIn_)
		hipFree(d_imgIn_);
	if(d_imgOut_)
		hipFree(d_imgOut_);
}

// do boxfilter on separable two dimension accumulation
// process row
// 加运算比移位运算优先级高
__device__ void d_boxfilter_x(float *Out, float *imgIn, int wid, int hei, int filterR)
{
	float scale = 1.0f / (float)((filterR << 1) + 1);
	//float scale = 0.0322581;
	float t;

	// do the left edge
	t = imgIn[0] * filterR;
	for(int x = 0; x < (filterR + 1); x++)
	{
		t += imgIn[x];
	}

	Out[0] = t * scale;

	for(int x = 1; x < (filterR + 1); x++)
	{
		t += imgIn[x + filterR];
		t -= imgIn[0];
		Out[x] = t * scale;
	}

	// main loop
	for(int x = (filterR + 1); x < (wid - filterR); x++)
	{
		t += imgIn[x + filterR];
		t -= imgIn[x - filterR - 1];
		Out[x] = t * scale;
	}

	// do the right edge
	for(int x = (wid - filterR); x < wid; x++)
	{
		t += imgIn[wid - 1];
		t -= imgIn[x - filterR - 1];
		Out[x] = t *  scale;
	}
}

// process column
__device__ void d_boxfilter_y(float *imgOut,float *imgIn, int wid, int hei, int filterR)
{
	float scale = 1.0f / (float)((filterR << 1) + 1);
	//float scale = 0.0322581;

	float t;

	// do the upper edge
	t = imgIn[0] * filterR;
	for(int y = 0; y < (filterR + 1); y++)
	{
		t += imgIn[y * wid];
	}

	imgOut[0] = 1.0 * t * scale;

	for(int y = 1; y < (filterR + 1); y++)
	{
		t += imgIn[(y + filterR) * wid];
		t -= imgIn[0];
		imgOut[y * wid] = t * scale;
	}

	// main loop
	for(int y = filterR + 1; y < hei - filterR; y++)
	{
		t += imgIn[(y + filterR) * wid];
		t -= imgIn[(y - filterR - 1) * wid];
		imgOut[y * wid] = t * scale;
	}

	// do the bottom dege
	for(int y = hei - filterR; y < hei; y++)
	{
		t += imgIn[(hei - 1) * wid];
		t -= imgIn[(y - filterR - 1) * wid];
		imgOut[y * wid] = t * scale;
	}
}

__global__ void d_boxfilter_x_global(float *Out, float *In, int wid, int hei, int filterR)
{
	unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;
	if( y >= hei)
		return ;
	d_boxfilter_x(&Out[y * wid], &In[y * wid], wid, hei, filterR);
}

__global__ void d_boxfilter_y_global(float *Out, float *In, int wid, int hei, int filterR)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= wid)
		return ;

	d_boxfilter_y(&Out[x], &In[x], wid, hei, filterR);
}

__global__ void elemwiseSub_kernel(float *out, float *inA, float *inB, int wid, int hei)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;

	if(idx >= wid || idy >= hei)
		return ;

	int offset = idy * wid + idx;
	out[offset] = inA[offset] - inB[offset];
}

void BFilter::boxfilter(float *d_imgOut, float *d_imgIn, int wid, int hei, int filterR)
{
	int nthreads = 512;

	float *d_temp;
	cudaCheckError(hipMalloc((void **)&d_temp, sizeof(float) * wid * hei));

	cudaCheckError(hipMemset(d_temp, 0, sizeof(float) * wid * hei));

	dim3 threadPerBlock(nthreads, 1);
	dim3 blockPerGrid;
	blockPerGrid.x = (hei + threadPerBlock.x - 1) / threadPerBlock.x;
	blockPerGrid.y = 1;

	// only one iteration
	d_boxfilter_x_global<<<blockPerGrid, threadPerBlock>>>(d_temp, d_imgIn, wid, hei, filterR);
	cout << hipGetErrorString(hipPeekAtLastError()) << endl;

	blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
	blockPerGrid.y = 1;
	d_boxfilter_y_global<<<blockPerGrid, threadPerBlock>>>(d_imgOut, d_temp, wid, hei, filterR);
	cout << hipGetErrorString(hipPeekAtLastError()) << endl;

	cudaCheckError(hipDeviceSynchronize());

	if(d_temp)
		hipFree(d_temp);
}

void BFilter::boxfilterTest(float *imgOut, float *imgIn, int wid, int hei, int filterR)
{
	int nthreads = 512;

	float *d_temp;
	cudaCheckError(hipMalloc((void **)&d_temp, sizeof(float) * wid * hei));

	cudaCheckError(hipMemset(d_temp, 0, sizeof(float) * wid * hei));

	float *d_imgIn, *d_imgOut;
	cudaCheckError(hipMalloc((void **)&d_imgIn, sizeof(float) * wid * hei));
	cudaCheckError(hipMemcpy(d_imgIn, imgIn, sizeof(float) * wid * hei, hipMemcpyHostToDevice));
	cudaCheckError(hipMalloc((void **)&d_imgOut, sizeof(float) * wid * hei));

	dim3 threadPerBlock(nthreads, 1);
	dim3 blockPerGrid;
	blockPerGrid.x = (hei + threadPerBlock.x - 1) / threadPerBlock.x;
	blockPerGrid.y = 1;

	// only one iteration
	d_boxfilter_x_global<<<blockPerGrid, threadPerBlock>>>(d_temp, d_imgIn, wid, hei, filterR);
	//d_boxfilter_x_global<<<blockPerGrid, threadPerBlock>>>(d_imgOut_, d_imgIn_, wid, hei, filterR);
	cout << hipGetErrorString(hipPeekAtLastError()) << endl;

	blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
	blockPerGrid.y = 1;
	d_boxfilter_y_global<<<blockPerGrid, threadPerBlock>>>(d_imgOut, d_temp, wid, hei, filterR);
	//d_boxfilter_y_global<<<blockPerGrid, threadPerBlock>>>(d_imgOut_, d_imgIn_, wid, hei, filterR);

	cout << hipGetErrorString(hipPeekAtLastError()) << endl;


	cudaCheckError(hipMemcpy(imgOut, d_imgOut, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));
	cudaCheckError(hipDeviceSynchronize());

	if(d_temp)
		hipFree(d_temp);
	if(d_imgIn)
		hipFree(d_imgIn);
	if(d_imgOut)
		hipFree(d_imgOut);
}

/*
void BFilter::boxfilterTest(float *imgOut, float *imgIn, int wid, int hei, int filterR)
{
	int nthreads = 512;

	float *d_temp;
	cudaCheckError(hipMalloc((void **)&d_temp, sizeof(float) * wid * hei));

	cudaCheckError(hipMemset(d_temp, 0, sizeof(float) * wid * hei));

	cudaCheckError(hipMemcpy(d_imgIn_, imgIn, sizeof(float) * wid * hei, hipMemcpyHostToDevice));

	dim3 threadPerBlock(nthreads, 1);
	dim3 blockPerGrid;
	blockPerGrid.x = (hei + threadPerBlock.x - 1) / threadPerBlock.x;
	blockPerGrid.y = 1;

	// only one iteration
	d_boxfilter_x_global<<<blockPerGrid, threadPerBlock>>>(d_temp, d_imgIn_, wid, hei, filterR);
	//d_boxfilter_x_global<<<blockPerGrid, threadPerBlock>>>(d_imgOut_, d_imgIn_, wid, hei, filterR);
	cout << hipGetErrorString(hipPeekAtLastError()) << endl;

	blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
	blockPerGrid.y = 1;
	d_boxfilter_y_global<<<blockPerGrid, threadPerBlock>>>(d_imgOut_, d_temp, wid, hei, filterR);
	//d_boxfilter_y_global<<<blockPerGrid, threadPerBlock>>>(d_imgOut_, d_imgIn_, wid, hei, filterR);

	cout << hipGetErrorString(hipPeekAtLastError()) << endl;


	cudaCheckError(hipMemcpy(imgOut, d_imgOut_, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));
	cudaCheckError(hipDeviceSynchronize());

	if(d_temp)
		hipFree(d_temp);
}
*/

// Version 1 : not consider about the global memory coalesce
void TScale::twoscaleTest(float *imgOutA, float *imgOutB, float *imgIn, int wid, int hei, int filterR)
{
	float *d_imgIn, *d_imgOutA, *d_imgOutB;
	cudaCheckError(hipMalloc((void **)&d_imgIn, sizeof(float) * wid * hei));
	cudaCheckError(hipMemcpy(d_imgIn, imgIn, sizeof(float) * wid * hei, hipMemcpyHostToDevice));
	cudaCheckError(hipMalloc((void **)&d_imgOutA, sizeof(float) * wid * hei));
	cudaCheckError(hipMalloc((void **)&d_imgOutB, sizeof(float) * wid * hei));

	// get the low pass coefficients
	boxfilter(d_imgOutB, d_imgIn, wid, hei, filterR);

	// get the high pass coefficients
	dim3 threadPerBlock(BLK_SZ, BLK_SZ);
	dim3 blockPerGrid;
	blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
	blockPerGrid.y = (hei + threadPerBlock.y - 1) / threadPerBlock.y;

	elemwiseSub_kernel<<<blockPerGrid, threadPerBlock>>>(d_imgOutA, d_imgIn, d_imgOutB, wid, hei);
	cout << hipGetErrorString(hipPeekAtLastError()) << endl;

	cudaCheckError(hipMemcpy(imgOutB, d_imgOutB, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));
	cudaCheckError(hipMemcpy(imgOutA, d_imgOutA, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));

	hipDeviceSynchronize();
}
