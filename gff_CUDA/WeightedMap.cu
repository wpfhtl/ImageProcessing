#include "hip/hip_runtime.h"
#include "WeightedMap.h"

#define GaussW 11
#define GUIRAD_D 10
#define GUIEPS_D 0.3
#define GUIRAD_B 45
#define GUIEPS_B 0.3

WMap::WMap(int wid, int hei, int lr, int gr) : GFilter(wid, hei)
{
    int lw = (lr << 1) + 1;
    int gw = (gr << 1) + 1;
    cudaCheckErrors(hipMalloc((void **)&d_lap_, sizeof(float) * lw * lw));
    cudaCheckErrors(hipMalloc((void **)&d_gau_, sizeof(float) * gw));
    cudaCheckErrors(hipMalloc((void **)&d_tempE_, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMalloc((void **)&d_tempF_, sizeof(float) * wid * hei));
}


WMap::~WMap()
{
    if(d_lap_)
        hipFree(d_lap_);
    if(d_gau_)
        hipFree(d_gau_);
    if(d_tempE_)
        hipFree(d_tempE_);
    if(d_tempF_)
        hipFree(d_tempF_);
}

// do absolute laplacian filter based on shared memory
__global__ void laplacianAbs_kernel(float *out, float *in, int wid, int hei, const float * __restrict__ filter, int lr)
{
    int x0 = threadIdx.x;
    int y0 = threadIdx.y;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if(idx >= wid || idy >= hei)
        return ;

    extern __shared__ float shareMem[];

    //int TILEW = blockDim.x + 2 * lr;
    int TILEW = BLK_SZ + 2 * lr;

    int x, y;
    // copy data from global memory to shared memory, zero extends the border
    // case 1 : upper left
    x = idx - lr;
    y = idy - lr;
    if(x < 0 || y < 0)
        shareMem[INDX(y0, x0, TILEW)] = 0;
    else
        shareMem[INDX(y0, x0, TILEW)] = in[INDX(y, x, wid)];

    // case 2 : upper right
    x = idx + lr;
    y = idy - lr;
    if(x >= wid || y < 0)
        shareMem[INDX(y0, x0 + 2 * lr, TILEW)] = 0;
    else
        shareMem[INDX(y0, x0 + 2 * lr, TILEW)] = in[INDX(y, x, wid)];

    // case 3 : lower left
    x = idx - lr;
    y = idy + lr;
    if(x < 0 || y >= hei)
        shareMem[INDX(y0 + 2 * lr, x0, TILEW)] = 0;
    else
        shareMem[INDX(y0 + 2 * lr, x0, TILEW)] = in[INDX(y, x, wid)];

    // case 4 : lower right
    x = idx + lr;
    y = idy + lr;
    if(x >= wid || y >= hei)
        shareMem[INDX(y0 + 2 * lr, x0 + 2 * lr, TILEW)] = 0;
    else
        shareMem[INDX(y0 + 2 * lr, x0 + 2 * lr, TILEW)] = in[INDX(y, x, wid)];

    __syncthreads();

    int lw = (lr << 1) + 1;
    float val = 0.f;
    for(int i = 0; i < lw; ++i)             // row
        for(int j = 0; j < lw; ++j)         // col
            val += shareMem[INDX(y0 + i, x0 + j, TILEW)] * filter[INDX(i, j, lw)];

    // obtain the absolute value
    out[INDX(idy, idx, wid)] = val >= 0 ? val : -val;
}

// do separable gaussian filter based on CUDA
__global__ void gaussfilterRow_kernel(float *out, float *in, int wid, int hei, const float * __restrict__ filter, int gr)
{
    int x0 = threadIdx.x;
    int y0 = threadIdx.y;

    int idx = blockDim.x * blockIdx.x + x0;
    int idy = blockDim.y * blockIdx.y + y0;

    if(idx >= wid || idy >= hei)
        return ;

    extern __shared__ float shareMem[];

    int x, y;
    int TILEW = BLK_SZ + 2 * gr;
    // case 1 : left apron
    x = idx - gr;
    y = idy;
    if(x < 0)
        shareMem[INDX(y0, x0, TILEW)] = 0;
    else
        shareMem[INDX(y0, x0, TILEW)] = in[INDX(y, x, wid)];

    // case 2 : right apron
    x = idx + gr;
    y = idy;
    if(x >= wid)
        shareMem[INDX(y0, x0 + 2 * gr, TILEW)] = 0;
    else
        shareMem[INDX(y0, x0 + 2 * gr, TILEW)] = in[INDX(y, x, wid)];

    __syncthreads();

    float val = 0.f;
#pragma unrool
    for(int i = 0; i < GaussW; i++)
        val += __fmul_rd(shareMem[INDX(y0, x0 + i, TILEW)], filter[i]);

    out[INDX(idy, idx, wid)] = val;
}

__global__ void gaussfilterCol_kernel(float *out, float *in, int wid, int hei, float const * __restrict__ filter, int filterR)
{
    int x0 = threadIdx.x;
    int y0 = threadIdx.y;

    int idx = blockDim.x * blockIdx.x + x0;
    int idy = blockDim.y * blockIdx.y + y0;

    if (idx >= wid || idy >= hei)
        return;

    //__shared__ float shareMem[ * BLK_SZ];
    extern __shared__ float shareMem[];

    int x, y;
    // case 1 : top apron
    y = idy - filterR;
    x = idx;
    if(y < 0)
        shareMem[INDX(y0, x0, BLK_SZ)] = 0;
    else
        shareMem[INDX(y0, x0, BLK_SZ)] = in[INDX(y, x, wid)];

    // case 2 : bottom apron
    y = idy + filterR;
    x = idx;
    if(y >= hei)
        shareMem[INDX(y0 + 2 * filterR, x0, BLK_SZ)] = 0;
    else
        shareMem[INDX(y0 + 2 * filterR, x0, BLK_SZ)] = in[INDX(y, x, wid)];

    __syncthreads();

    float val = 0.f;
#pragma unroll
    for(int i = 0; i < GaussW; ++i)
        //val += shareMem[INDX(y0 + i, x0, BLK_SZ)] * filter[i];
        val += __fmul_rd(shareMem[INDX(y0+i, x0, BLK_SZ)], filter[i]);

    out[INDX(idy, idx, wid)] = val;
}

// comparasion kernel
__global__ void comparison_kernel(float *outA, float *outB, float *inA, float *inB, int wid, int hei)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = idx + idy * wid;

    if(idx >= wid || idy >= hei)
        return ;

    int val = (inA[offset] >= inB[offset]) ? 1 : 0;
    outA[offset] = val;
    outB[offset] = 1 - val;
}

// laplacian filter
void WMap::laplacianAbs(float *d_imgOut, float *d_imgIn, int wid, int hei, int lr)
{
    int lw = lr * 2 + 1;
    float *lapfilter = new float [lw * lw];

    hipStream_t st;
    cudaCheckErrors(hipStreamCreate(&st));

    // 3 * 3 laplacian filter
    lapfilter[0] = -1;      lapfilter[1] = -1;      lapfilter[2] = -1;
    lapfilter[3] = -1;      lapfilter[4] = 8;      lapfilter[5] = -1;
    lapfilter[6] = -1;      lapfilter[7] = -1;      lapfilter[8] = -1;

    // copy laplacian filter from host to device
    cudaCheckErrors(hipMemcpy(d_lap_, lapfilter, sizeof(float) * lw * lw, hipMemcpyHostToDevice));

    // do kernel on GPU
    dim3 threadPerBlock(BLK_SZ, BLK_SZ);
    dim3 blockPerGrid;
    blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
    blockPerGrid.y = (hei + threadPerBlock.y - 1) / threadPerBlock.y;

    // launch kernel function
    int TileW = BLK_SZ + 2 * lr;
    laplacianAbs_kernel<<<blockPerGrid, threadPerBlock, sizeof(float) * TileW * TileW, st>>>(d_imgOut, d_imgIn, wid, hei, d_lap_, lr);

    //cout << "Laplacian filter : " << hipGetErrorString(hipPeekAtLastError()) << endl;

    if(lapfilter)
        delete [] lapfilter;

    cudaCheckErrors(hipStreamDestroy(st));
}

void WMap::laplacianAbsTest(float *imgOut, float *imgIn, int wid, int hei, int lr)
{
    float *d_imgIn, *d_imgOut;

    cudaCheckErrors(hipMalloc((void **)&d_imgIn, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMemcpy(d_imgIn, imgIn, sizeof(float) * wid * hei, hipMemcpyHostToDevice));

    cudaCheckErrors(hipMalloc((void **)&d_imgOut, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMemset(d_imgOut, 0, sizeof(float) * wid * hei));

    laplacianAbs(d_imgOut, d_imgIn, wid, hei, lr);

    cudaCheckErrors(hipMemcpy(imgOut, d_imgOut, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    hipFree(d_imgIn);
    hipFree(d_imgOut);
}

void WMap::gaussian(float *d_imgOut, float *d_imgIn, int wid, int hei, int gr, int gsigma)
{
    int filterW = 2 * gr + 1;
    float *filter = new float [filterW];

    hipStream_t st;
    cudaCheckErrors(hipStreamCreate(&st));

    // generate row filter :
    filter[0] = -0.0663;    filter[1] = -0.0794;    filter[2] = -0.0914;
    filter[3] = -0.1010;    filter[4] = -0.1072;    filter[5] = -0.1094;
    filter[6] = -0.1072;    filter[7] = -0.1010;    filter[8] = -0.0914;
    filter[9] = -0.0794;    filter[10] = -0.0663;


    // copy filter data from host to device
    cudaCheckErrors(hipMemcpy(d_gau_, filter, sizeof(float) * filterW, hipMemcpyHostToDevice));

    // prepare needed memory on device
    float *d_temp;
    cudaCheckErrors(hipMalloc((void **)&d_temp, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMemset(d_temp, 0, sizeof(float) * wid * hei));

    int TileW = BLK_SZ + 2 * gr;

    dim3 threadPerBlock(BLK_SZ, BLK_SZ);
    dim3 blockPerGrid;
    blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
    blockPerGrid.y = (hei + threadPerBlock.y - 1) / threadPerBlock.y;

    gaussfilterRow_kernel<<<blockPerGrid, threadPerBlock, sizeof(float) * TileW * BLK_SZ, st>>>(d_temp, d_imgIn, wid, hei, d_gau_, gr);
    //cout << "In gaussian filter ROW part : " << hipGetErrorString(hipPeekAtLastError()) << endl;

    gaussfilterCol_kernel<<<blockPerGrid, threadPerBlock, sizeof(float) * TileW * BLK_SZ, st>>>(d_imgOut, d_temp, wid, hei, d_gau_, gr);
    //cout << "In gaussian filter COL part : " << hipGetErrorString(hipPeekAtLastError()) << endl;

    hipStreamDestroy(st);

    if(d_temp)
        hipFree(d_temp);
    if(filter)
        delete [] filter;
}

void WMap::gaussianTest(float *imgOut, float *imgIn, int wid, int hei, int gr, int gsigma)
{
    float *d_imgIn, *d_imgOut;

    cudaCheckErrors(hipMalloc((void **)&d_imgIn, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMemcpy(d_imgIn, imgIn, sizeof(float) * wid * hei, hipMemcpyHostToDevice));

    cudaCheckErrors(hipMalloc((void **)&d_imgOut, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMemset(d_imgOut, 0, sizeof(float) * wid * hei));

    gaussian(d_imgOut, d_imgIn, wid, hei, gr, 0.1);

    cudaCheckErrors(hipMemcpy(imgOut, d_imgOut, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    hipFree(d_imgIn);
    hipFree(d_imgOut);
}

// do the saliency map generation
void WMap::saliencymapTest(float *imgOut, float *imgIn, int wid, int hei, int lr, int gr, double gsigma)
{
    float *d_imgIn, *d_imgOut;

    cudaCheckErrors(hipMalloc((void **)&d_imgIn, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMemcpy(d_imgIn, imgIn, sizeof(float) * wid * hei, hipMemcpyHostToDevice));

    cudaCheckErrors(hipMalloc((void **)&d_imgOut, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMemset(d_imgOut, 0, sizeof(float) * wid * hei));

    laplacianAbs(d_imgOut, d_imgIn, wid, hei, lr);
    gaussian(d_imgOut, d_imgOut, wid, hei, gr, gsigma);

    cudaCheckErrors(hipMemcpy(imgOut, d_imgOut, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    hipFree(d_imgIn);
    hipFree(d_imgOut);
}

void WMap::weightedmap(float *d_imgOutA, float *d_imgOutB, float *d_imgOutC, float *d_imgOutD, float *d_imgInA,
                       float *d_imgInB, int wid, int hei, int lr, int gr, int gsigma, int guir, double eps)
{
    laplacianAbs(d_tempE_, d_imgInA, wid, hei, lr);
    laplacianAbs(d_tempF_, d_imgInB, wid, hei, lr);

    //gaussian(d_imgOutA, d_tempA_, wid, hei, gr, gsigma);
    //gaussian(d_imgOutB, d_tempB_, wid, hei, gr, gsigma);
    gaussian(d_tempE_, d_tempE_, wid, hei, gr, gsigma);
    gaussian(d_tempF_, d_tempF_, wid, hei, gr, gsigma);

    dim3 threadPerBlock(BLK_SZ, BLK_SZ);
    dim3 blockPerGrid;
    blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
    blockPerGrid.y = (hei + threadPerBlock.y - 1) / threadPerBlock.y;
    comparison_kernel<<<blockPerGrid, threadPerBlock>>>(d_tempE_, d_tempF_, d_tempE_, d_tempF_, wid, hei);
    //comparison_kernel<<<blockPerGrid, threadPerBlock>>>(d_imgOutA, d_imgOutB, d_tempE_, d_tempF_, wid, hei);

    /*
    GFilter gf(wid, hei);
    gf.guidedfilter(d_imgOutA, d_imgInA, d_tempE_, wid, hei, guir, eps);
    gf.guidedfilter(d_imgOutB, d_imgInB, d_tempF_, wid, hei, guir, eps);
    */
    guidedfilter(d_imgOutA, d_imgInA, d_tempE_, wid, hei, GUIRAD_D, GUIEPS_D);
    guidedfilter(d_imgOutB, d_imgInA, d_tempE_, wid, hei, GUIRAD_B, GUIEPS_B);
    guidedfilter(d_imgOutC, d_imgInB, d_tempF_, wid, hei, GUIRAD_D, GUIEPS_D);
    guidedfilter(d_imgOutD, d_imgInB, d_tempF_, wid, hei, GUIRAD_B, GUIEPS_B);
    //guidedfilter(d_imgOutB, d_imgInB, d_tempF_, wid, hei, guir * 2, eps / 100);

    hipDeviceSynchronize();
}

void WMap::weightedmapTest(float *imgOutA, float *imgOutB, float *imgInA, float *imgInB,
                           int wid, int hei, int lr, int gr, int gsigma, int guir, double eps)
{
    hipEvent_t cudaStart, cudaStop;

    float *d_imgInA, *d_imgOutA, *d_imgInB, *d_imgOutB;
    cudaCheckErrors(hipMalloc((void **)&d_imgInA, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMalloc((void **)&d_imgInB, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMalloc((void **)&d_imgOutA, sizeof(float) * wid * hei));
    cudaCheckErrors(hipMalloc((void **)&d_imgOutB, sizeof(float) * wid * hei));

    cudaCheckErrors(hipMemcpy(d_imgInA, imgInA, sizeof(float) * wid * hei, hipMemcpyHostToDevice));
    cudaCheckErrors(hipMemcpy(d_imgInB, imgInB, sizeof(float) * wid * hei, hipMemcpyHostToDevice));

    hipEventCreate(&cudaStart);
    hipEventCreate(&cudaStop);

    hipEventRecord(cudaStart, 0);

    //weightedmap(d_imgOutA, d_imgOutB, d_imgInA, d_imgInB, wid, hei, lr, gr, gsigma, guir, eps);
    weightedmap(d_imgOutA, d_imgOutB, d_imgOutA, d_imgOutB, d_imgInA, d_imgInB, wid, hei, lr, gr, gsigma, guir, eps);

    hipEventRecord(cudaStop, 0);
    hipEventSynchronize(cudaStop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, cudaStart, cudaStop);
    cout << "Weighted Map GPU Time (no memory copy) : " << elapsedTime << " ms" << endl;

    cudaCheckErrors(hipMemcpy(imgOutA, d_imgOutA, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));
    cudaCheckErrors(hipMemcpy(imgOutB, d_imgOutB, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));
}
